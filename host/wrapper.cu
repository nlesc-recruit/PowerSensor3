#include "hip/hip_runtime.h"
//#include <stdio.h>
//#include <chrono>
#include "wrapper.cuh"

//#include "PowerSensor.h"

int compare_arrays(float *c, float *d, int n);

void vec_add(float *c, float *a, float *b, int n) {
    for (int i=0; i<n; i++) {
        c[i] = a[i] + b[i];
    }
}

extern "C" __global__ void vec_add_kernel(float *c, float *a, float *b, int n) {
    //int i = blockIdx.x*threadIdx.x+blockDim.x;   // Oops! Something is not right here, please fix it!
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

namespace Wrapper {
int n = 5e7;
hipError_t err;

float *a = (float *) malloc(n * sizeof(float));
float *b = (float *) malloc(n * sizeof(float));
float *c = (float *) malloc(n * sizeof(float));
float *d = (float *) malloc(n * sizeof(float));
float *d_a; float *d_b; float *d_c;

void setup(void) {
    //printf("%d", n);
    //allocate arrays and fill them
    for (int i=0; i<n; i++) {
        a[i] = 1.0 / rand();
        b[i] = 1.0 / rand();
        c[i] = 0.0;
        d[i] = 0.0;
    }

    //allocate GPU memory
    err = hipMalloc((void **)&d_a, n*sizeof(float));
    if (err != hipSuccess) fprintf(stderr, "Error in hipMalloc d_a: %s\n", hipGetErrorString( err ));
    err = hipMalloc((void **)&d_b, n*sizeof(float));
    if (err != hipSuccess) fprintf(stderr, "Error in hipMalloc d_b: %s\n", hipGetErrorString( err ));
    err = hipMalloc((void **)&d_c, n*sizeof(float));
    if (err != hipSuccess) fprintf(stderr, "Error in hipMalloc d_c: %s\n", hipGetErrorString( err ));
    
    
    //powerSensor.mark("copy");
  
    //copy the input data to the GPU
    err = hipMemcpy(d_a, a, n*sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) fprintf(stderr, "Error in hipMemcpy host to device a: %s\n", hipGetErrorString( err ));
    err = hipMemcpy(d_b, b, n*sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) fprintf(stderr, "Error in hipMemcpy host to device b: %s\n", hipGetErrorString( err ));

    //zero the output array
    err = hipMemset(d_c, 0, n*sizeof(float));
    if (err != hipSuccess) fprintf(stderr, "Error in hipMemset c: %s\n", hipGetErrorString( err ));
}


void executeKernel(void) {    
    //setup the grid and thread blocks
    int block_size = 1024;                          //thread block size
    int nblocks = int(ceilf(n/(float)block_size));  //problem size divided by thread block size rounded up
    dim3 grid(nblocks, 1);
    dim3 threads(block_size, 1, 1);

    //measure the GPU function
    hipDeviceSynchronize();
    vec_add_kernel<<<grid, threads>>>(d_c, d_a, d_b, n);
    hipDeviceSynchronize();

    err = hipGetLastError();
    if (err != hipSuccess) fprintf(stderr, "Error during kernel launch vec_add_kernel: %s\n", hipGetErrorString( err ));
}    
    //powerSensor.mark("done");
void cleanUp(void) {
    //check to see if all went well

    //copy the result back to host memory
    err = hipMemcpy(d, d_c, n*sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) fprintf(stderr, "Error in hipMemcpy device to host c: %s\n", hipGetErrorString( err ));

    //powerSensor.dump(0);

    //clean up
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);
    free(d);

}
}


int compare_arrays(float *a1, float *a2, int n) {
    int errors = 0;
    int print = 0;

    for (int i=0; i<n; i++) {

        if (isnan(a1[i]) || isnan(a2[i])) {
            errors++;
            if (print < 10) {
                print++;
                fprintf(stderr, "Error NaN detected at i=%d,\t a1= %10.7e \t a2= \t %10.7e\n",i,a1[i],a2[i]);
            }
        }

        unsigned int int_a1 = *(unsigned int *)(a1+i);
        unsigned int int_a2 = *(unsigned int *)(a2+i);
        unsigned int dist = (unsigned int)0;
        if (int_a1 > int_a2) {
            dist = int_a1 - int_a2;
        } else {
            dist = int_a2 - int_a1;
        }
        if (dist > 0) {
            errors++;
            if (print < 10) {
                print++;
                fprintf(stderr, "Error detected at i=%d, \t a1= \t %10.7e \t a2= \t %10.7e \t ulp_dist=\t %u\n",i,a1[i],a2[i],dist);
            }
        }

    }

    return errors;
}
